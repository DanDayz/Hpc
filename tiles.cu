#include "hip/hip_runtime.h"
# include <bits/stdc++.h>
# include <hip/hip_runtime.h>

#define TILE_WIDTH 32 //(TITLE_WIDTH = BLOCKSIZE)

using namespace std;
// ::::::::::::::::::::::::::::::::::::::::::GPU::::::::::::::::::::::::::::::::

__global__ void KernelNormalMul(int *Mat1,int *Mat2,int *Mat3,int m,int n,int p){
  int j = threadIdx.y + blockDim.y * blockIdx.y; // row
  int i = threadIdx.x + blockDim.x * blockIdx.x; // col

  if((j<m) && (i<p)){
    int value=0;
    for(int k=0;k<n;++k){
      value+=Mat1[n*j+k]*Mat2[p*k+i];
    }
    Mat3[p*j+i]=value;
  }
}


__global__ void  KernelTilesMul(int *Mat1,int *Mat2,int *Mat3,int rowM1,int colM1,int colM2){

  __shared__ int Mds[TILE_WIDTH][TILE_WIDTH];
  __shared__ int Nds[TILE_WIDTH][TILE_WIDTH];

  int bx = blockIdx.x;
  int by = blockIdx.y;
  int tx = threadIdx.x;
  int ty = threadIdx.y;

  int row = by * TILE_WIDTH + ty;
  int col = bx * TILE_WIDTH + tx;

  int Pvalue = 0;


  for(int k = 0; k < (colM1+TILE_WIDTH-1)/(TILE_WIDTH); ++k){

    if(k*TILE_WIDTH + tx < colM1 && row < rowM1){
      Mds[ty][tx] = Mat1[row*colM1 + k*TILE_WIDTH + tx];
    }else{
      Mds[ty][tx] = 0;
    }
    if(k*TILE_WIDTH + ty < colM1 && col < colM2){
      Nds[ty][tx] = Mat2[(k*TILE_WIDTH + ty) * colM2 + col];
    }else{
      Nds[ty][tx] =0;
    }

    __syncthreads();

    for(int k = 0; k < TILE_WIDTH; ++k){
      Pvalue += Mds[ty][k] * Nds[k][tx];
    }
    __syncthreads();
  }

  if (row < rowM1 && col < colM2){
    Mat3[row*colM2+col] = Pvalue;
  }

}



void d_MatrixMult(int *Mat1,int *Mat2,int *Mat3,int rowM1,int colM1,int colM2, int op ){
  int * d_Mat1;
  int * d_Mat2;
  int * d_Mat3;
  float Blocksize=TILE_WIDTH; // Bloque de 2 dimensiones 32*32=256  número de blokes= 1024 (1024/256=4)
  int size1=rowM1*colM1;
  int size2=colM1*colM2;
  int size3=rowM1*colM2;

  // 1. Separamos memoria en el device
  hipMalloc(&d_Mat1,size1*sizeof(int));
  hipMalloc(&d_Mat2,size2*sizeof(int));
  hipMalloc(&d_Mat3,size3*sizeof(int));

  // 2. Copiamos el valor de las variables de host a las variables del device.
  hipMemcpy(d_Mat1, Mat1,size1*sizeof(int),hipMemcpyHostToDevice);
  hipMemcpy(d_Mat2, Mat2,size2*sizeof(int),hipMemcpyHostToDevice);
  // 3. Lógica de bloques e hilos, elementos para realizar la parelelización.
  dim3 dimGrid(ceil(colM2/Blocksize),ceil(rowM1/Blocksize),1);
  //dim3 dimGrid((m+Blocksize-1)/Blocksize,(p+Blocksize-1)/Blocksize,1);
  dim3 dimBlock(Blocksize,Blocksize,1);
  // 4. Invocación del kernel (invoción del host, ejecutadas en el device), <<<<#dimGrid,#dimBlock>>>
  if(op==1){KernelNormalMul<<<dimGrid,dimBlock>>>(d_Mat1,d_Mat2,d_Mat3,rowM1,colM1,colM2);}else{
    KernelTilesMul<<<dimGrid,dimBlock>>>(d_Mat1,d_Mat2,d_Mat3,rowM1,colM1,colM2);
  }
  // 5. Copiamos el resultado para mostrar en el I/O del host.
  hipMemcpy (Mat3,d_Mat3,size3*sizeof(int),hipMemcpyDeviceToHost);
  // 6. Liberamos memoria.
  hipFree(d_Mat3);
}

// :::::::::::::::::::::::::::::::::::::::Normal::::::::::::::::::::::::::::::::


void h_Mul_Mat(int *Mat1,int *Mat2, int *Mat3,int m,int n,int p){

  for(int i=0;i<m;i++){
    for(int j=0;j<p;j++){
      int value=0;
      for(int k=0;k<n;k++){
        value+=Mat1[n*i+k]*Mat2[p*k+j];
      }
      Mat3[p*i+j]=value;
    }
  }
}

void llena_mat(int *Mat, int Value,int m,int n){// ver matriz como vector serial.
  int size=n*m; // matriz lineal
  for(int i =0 ; i<size ; i++){
    Mat[i]=Value;
  }
}

void mostrar_mat(int *Mat,int m,int n){//
  int size=n*m; // matriz lineal
  for (int i=0;i<size;i++) {
    if(i%n==0 && n!=0){
      cout<<endl;
    }
    cout<<"["<<Mat[i]<<"] ";
  }
  cout<<endl;
}


int check_mat(int *Mat1,int *Mat2,int m,int p){
  for(int i=0; i<(m*p);++i){
    if(Mat1[i]!=Mat2[i]){
      cout<<"Error, Las matrices no son iguales"<<endl;
      return 0;
    }
  }
  cout<<"Las Matrices son iguales"<<endl;
  return 0;
}



// :::::::::::::::::::::::::::::::::::Clock Function::::::::::::::::::::::::::::

double diffclock(clock_t clock1,clock_t clock2){
  double diffticks=clock2-clock1;
  double diffms=(diffticks)/(CLOCKS_PER_SEC/1); //  /1000 mili
  return diffms;
}

// :::::::::::::::::::::::::::::::::::::::Main::::::::::::::::::::::::::::::::.

int main(){
  double T1,T2,T3; // variables de tiempo

  int rowM1=2;
  int colM1=4;
  int colM2=4;
  int *Mat1 = (int*)malloc((rowM1*colM1)*sizeof(int));
  int *Mat2 = (int*)malloc((colM1*colM2)*sizeof(int));
  int *Mat3 = (int*)malloc((rowM1*colM2)*sizeof(int));
  int *Mat4 = (int*)malloc((rowM1*colM2)*sizeof(int));
  int *Mat5 = (int*)malloc((rowM1*colM2)*sizeof(int));

  llena_mat(Mat1,1,rowM1,colM1);
  llena_mat(Mat2,1,colM1,colM2);

  clock_t start = clock();
  h_Mul_Mat(Mat1,Mat2,Mat3,rowM1,colM1,colM2);
  clock_t end = clock();
  T1=diffclock(start,end);
  cout <<"Tiempo secuencial: "<<T1<<endl;
  mostrar_mat(Mat3,rowM1,colM2);
  clock_t start2 = clock();
  d_MatrixMult(Mat1,Mat2,Mat4,rowM1,colM1,colM2,1); // paralelo
  clock_t end2 = clock();
  mostrar_mat(Mat4,rowM1,colM2);
  T2=diffclock(start2,end2);
  cout <<"Tiempo Paralelo: "<<T2<<endl;
  cout<<"Aceleración lograda: "<<T1/T2<<endl;

  check_mat(Mat3,Mat4,rowM1,colM2);


  clock_t start3 = clock();
  d_MatrixMult(Mat1,Mat2,Mat5,rowM1,colM1,colM2,2); // tiles
  mostrar_mat(Mat5,rowM1,colM2);
  clock_t end3 = clock();
  T3=diffclock(start3,end3);

  cout <<"Tiempo Paralelo con Tiles: "<<T3<<endl;
  cout<<"Aceleración lograda Respecto a el tiempo paralelo: "<<T2/T3<<endl;

  check_mat(Mat4,Mat5,rowM1,colM2);

  free(M1);
  free(M2);
  free(M3);
  free(M4);
  free(M5);

  return 0;
}

// http://www.techdarting.com/2014/03/matrix-multiplication-in-cuda-using.html
