# include <bits/stdc++.h>
# include <hip/hip_runtime.h>

#define TILE_WIDTH 32

using namespace std;
// ::::::::::::::::::::::::::::::::::::::::::GPU::::::::::::::::::::::::::::::::

__global__ void KernelNormalMul(int *Mat1,int *Mat2,int *Mat3,int m,int n,int p){
  int j = threadIdx.y + blockDim.y * blockIdx.y; // cols
  int i = threadIdx.x + blockDim.x * blockIdx.x; // row

  if((i<m) && (j<p)){
    int value=0;
    for(int k=0;k<n;k++){
      value+=Mat1[n*i+k]*Mat2[p*k+j];
    }
    Mat3[p*i+j]=value;
  }
}


__global__ void  KernelTilesMul(int *Mat1,int *Mat2,int *Mat3,int m,int n,int p){

      __shared__ int Mds[TILE_WIDTH][TILE_WIDTH];
      __shared__ int Nds[TILE_WIDTH][TILE_WIDTH];
      int bx = blockIdx.x;
      int by = blockIdx.y;
      int tx = threadIdx.x;
      int ty = threadIdx.y;
      int row = by * TILE_WIDTH + ty;
      int col = bx * TILE_WIDTH + tx;
      int Pvalue = 0;


        for(int k = 0; k < (m + TILE_WIDTH-1)/TILE_WIDTH; ++k){

          if(k*TILE_WIDTH + tx < n && row < m){
            Mds[ty][tx] = Mat1[row*n + k*TILE_WIDTH + tx];
          }else{
            Mds[ty][tx] = 0;
          }
          if(k*TILE_WIDTH + threadIdx.y < n && col < p){
            Nds[ty][tx] = Mat2[(k*TILE_WIDTH + ty) * p + col];
          }else{
            Nds[ty][tx] =0;
          }

        __syncthreads();

        for(int k = 0; k < TILE_WIDTH; ++k){
          Pvalue += Mds[ty][k] * Nds[k][tx];
        }
          __syncthreads();
      }

      if (row < m && col < p){
        Mat3[row*p+col] = Pvalue;
        }

}



void d_MatrixMult(int *Mat1,int *Mat2,int *Mat3,int m,int n,int p, int op ){
  int * d_Mat1;
  int * d_Mat2;
  int * d_Mat3;
  float Blocksize=32; // Bloque de 2 dimensiones 32*32=256  número de blokes= 1024 (1024/256=4)
  int size1=m*n;
  int size2=n*p;
  int size3=m*p;

  // 1. Separamos memoria en el device
  hipMalloc(&d_Mat1,size1*sizeof(int));
  hipMalloc(&d_Mat2,size2*sizeof(int));
  hipMalloc(&d_Mat3,size3*sizeof(int));

  // 2. Copiamos el valor de las variables de host a las variables del device.
  hipMemcpy(d_Mat1, Mat1,size1*sizeof(int),hipMemcpyHostToDevice);
  hipMemcpy(d_Mat2, Mat2,size2*sizeof(int),hipMemcpyHostToDevice);
  // 3. Lógica de bloques e hilos, elementos para realizar la parelelización.
  dim3 dimGrid(ceil(m/Blocksize),ceil(p/Blocksize),1);
  //dim3 dimGrid((m+Blocksize-1)/Blocksize,(p+Blocksize-1)/Blocksize,1);
  dim3 dimBlock(Blocksize,Blocksize,1);
  // 4. Invocación del kernel (invoción del host, ejecutadas en el device), <<<<#dimGrid,#dimBlock>>>
  if(op==1){KernelNormalMul<<<dimGrid,dimBlock>>>(d_Mat1,d_Mat2,d_Mat3,m,n,p);}else{
  	KernelTilesMul<<<dimGrid,dimBlock>>>(d_Mat1,d_Mat2,d_Mat3,m,n,p);
  }
  // 5. Copiamos el resultado para mostrar en el I/O del host.
  hipMemcpy (Mat3,d_Mat3,size3*sizeof(int),hipMemcpyDeviceToHost);
  // 6. Liberamos memoria.
  hipFree(d_Mat3);
}

// :::::::::::::::::::::::::::::::::::::::Normal::::::::::::::::::::::::::::::::


void h_Mul_Mat(int *Mat1,int *Mat2, int *Mat3,int m,int n,int p){

  for(int i=0;i<m;i++){
    for(int j=0;j<p;j++){
      int value=0;
      for(int k=0;k<n;k++){
        value+=Mat1[n*i+k]*Mat2[p*k+j];
        }
			Mat3[p*i+j]=value;
    }
  }
}

void llena_mat(int *Mat, int Value,int m,int n){// ver matriz como chorizo grande
 int size=n*m; // matriz lineal
  for(int i =0 ; i<size ; i++){
          Mat[i]=Value;
      }
}

void mostrar_mat(int *Mat,int m,int n){// ver matriz como chorizo grande
 int size=n*m; // matriz lineal
  for (int i=0;i<size;i++) {
      if(i%n==0 && n!=0){
        cout<<endl;
      }
    cout<<"["<<Mat[i]<<"] ";
  }
  cout<<endl;
}


int check_mat(int *Mat1,int *Mat2,int m,int p){
  for(int i=0; i<(m*p);i++){
    if(Mat1[i]!=Mat2[i]){
      cout<<"Error, Las matrices no son iguales"<<endl;
      return 1;
    }
  }
  return 0;
}


// :::::::::::::::::::::::::::::::::::Clock Function::::::::::::::::::::::::::::

double diffclock(clock_t clock1,clock_t clock2){
    double diffticks=clock2-clock1;
    double diffms=(diffticks)/(CLOCKS_PER_SEC/1); //  /1000 mili
    return diffms;
}

// :::::::::::::::::::::::::::::::::::::::Main::::::::::::::::::::::::::::::::.

int main(){
  // Malloc   (Fila,Columna)
	double T1,T2,T3;

  int n=32; // columna Mat1, fila Mat2
  int m=64; // Fila Mat1 , Fila Mat3
  int p=64; // colunma Mat2m Columna Mat3
  int *Mat1 = (int*)malloc((m*n)*sizeof(int));
  int *Mat2 = (int*)malloc((n*p)*sizeof(int));
  int *Mat3 = (int*)malloc((m*p)*sizeof(int));
  int *Mat4 = (int*)malloc((m*p)*sizeof(int));
  int *Mat5 = (int*)malloc((m*p)*sizeof(int));

  llena_mat(Mat1,1,m,n);
  llena_mat(Mat2,2,n,p);

	clock_t start = clock();
  h_Mul_Mat(Mat1,Mat2,Mat3,m,n,p);
  clock_t end = clock();
  T1=diffclock(start,end);
  cout <<"Tiempo secuencial: "<<T1<<endl;
  //mostrar_mat(Mat3,m,p);
  clock_t start2 = clock();
  d_MatrixMult(Mat1,Mat2,Mat4,m,n,p,1); // paralelo
  clock_t end2 = clock();
  //mostrar_mat(Mat4,m,p);
  T2=diffclock(start2,end2);
  cout <<"Tiempo Paralelo: "<<T2<<endl;
  cout<<"Aceleración lograda: "<<T1/T2<<endl;

  if(check_mat(Mat3,Mat4,m,p)==0){
   cout<<"Matrices M1 Y M2 son iguales"<<endl;
  }

  clock_t start3 = clock();
  d_MatrixMult(Mat1,Mat2,Mat5,m,n,p,2); // tiles
  //mostrar_mat(Mat5,m,p);
  clock_t end3 = clock();
  T3=diffclock(start3,end3);

  cout <<"Tiempo Paralelo con Tiles: "<<T3<<endl;
	cout<<"Aceleración lograda Respecto a el tiempo paralelo: "<<T2/T3<<endl;

   if(check_mat(Mat4,Mat5,m,p)==0){
   cout<<"Matrices M2 Y M3 son iguales"<<endl;
  }

return 0;
}

// http://www.techdarting.com/2014/03/matrix-multiplication-in-cuda-using.html
