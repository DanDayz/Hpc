#include "hip/hip_runtime.h"
/* Daniel Diaz Giraldo

Restrictions
Mask = 5, Only works whit odd numbers and Mask size <= N _elements;
N_elements = defined by architecture from machine; (Femin-Maxwell....) in this case
i'm use a Kepler Arch; (the number of blocks that can support is around 2^31)

*/

#include <bits/stdc++.h>
#include <hip/hip_runtime.h>
#include <highgui.h>
#include <cv.h>

#define N_elements 32
#define Mask_size  3
#define TILE_SIZE  1024
#define BLOCK_SIZE 32

using namespace std;
using namespace cv;

__constant__ char Global_Mask[Mask_size*Mask_size];

__device__ unsigned char clamp(int value){
    if(value < 0)
        value = 0;
    else
        if(value > 255)
            value = 255;
    return  value;
}



__global__ void convolution2d_global_kernel(unsigned char *In,char *M, unsigned char *Out,int Mask_Width,int Rowimg,int Colimg){

   unsigned int row = blockIdx.y*blockDim.y+threadIdx.y;
   unsigned int col = blockIdx.x*blockDim.x+threadIdx.x;

   int Pvalue = 0;

   int N_start_point_row = row - (Mask_Width/2);
   int N_start_point_col = col - (Mask_Width/2);

   for(int i = 0; i < Mask_Width; i++){
       for(int j = 0; j < Mask_Width; j++ ){
        if((N_start_point_col + j >=0 && N_start_point_col + j < Rowimg)&&(N_start_point_row + i >=0 && N_start_point_row + i < Colimg)){
               Pvalue += In[(N_start_point_row + i)*Rowimg+(N_start_point_col + j)] * M[i*Mask_Width+j];
           }
       }
   }

   Out[row*Rowimg+col] = clamp(Pvalue);

}

__global__ void convolution2d_constant_kernel(unsigned char *In,char *M, unsigned char *Out,int Mask_Width,int Rowimg,int Colimg){

   unsigned int row = blockIdx.y*blockDim.y+threadIdx.y;
   unsigned int col = blockIdx.x*blockDim.x+threadIdx.x;

   int Pvalue = 0;

   int N_start_point_row = row - (Mask_Width/2);
   int N_start_point_col = col - (Mask_Width/2);

   for(int i = 0; i < Mask_Width; i++){
       for(int j = 0; j < Mask_Width; j++ ){
        if((N_start_point_col + j >=0 && N_start_point_col + j < Rowimg)&&(N_start_point_row + i >=0 && N_start_point_row + i < Colimg)){
               Pvalue += In[(N_start_point_row + i)*Rowimg+(N_start_point_col + j)] * Global_Mask[i*Mask_Width+j];
           }
       }
   }
   Out[row*Rowimg+col] = clamp(Pvalue);
}



__global__ void convolution2d_tiled_constant_kernel(unsigned char *In,char *M, unsigned char *Out,int Mask_Width,int Rowimg,int Colimg){

   unsigned int row = blockIdx.y*blockDim.y+threadIdx.y;
   unsigned int col = blockIdx.x*blockDim.x+threadIdx.x;
   __shared__ int Tile[(TILE_SIZE + Mask_size - 1)*2];

   int

   int Pvalue = 0;
   int N_start_point_row = row - (Mask_Width/2);
   int N_start_point_col = col - (Mask_Width/2);

   for(int i = 0; i < Mask_Width; i++){
       for(int j = 0; j < Mask_Width; j++ ){
        if((N_start_point_col + j >=0 && N_start_point_col + j < Rowimg)&&(N_start_point_row + i >=0 && N_start_point_row + i < Colimg)){
               Pvalue += In[(N_start_point_row + i)*Rowimg+(N_start_point_col + j)] * Global_Mask[i*Mask_Width+j];
           }
       }
   }
   Out[row*Rowimg+col] = clamp(Pvalue);
}

/*
__global__ void convolution1d_tiles_constant_kernel(int *In, int *Out){
  unsigned int index = blockIdx.x * blockDim.x + threadIdx.x; // Index 1d iterator.
  __shared__ int Tile[TILE_SIZE + Mask_size - 1];
  int nx = Mask_size/2;
  int halo_left_index  = (blockIdx.x - 1 ) * blockDim.x + threadIdx.x;
  if (threadIdx.x  >= blockDim.x - n ){
     Tile[threadIdx.x - (blockDim.x - n )] = (halo_left_index < 0) ? 0 : In[halo_left_index];
  }

  if(index<N_elements){Tile[n + threadIdx.x] = In[index];
  }else{Tile[n + threadIdx.x] = 0;}
  int halo_right_index = (blockIdx.x + 1 ) * blockDim.x + threadIdx.x;
  if (threadIdx.x < n) {
    Tile[n + blockDim.x + threadIdx.x]=  (halo_right_index >= N_elements) ? 0 : In[halo_right_index];
  }

  int Value = 0;
__syncthreads();
  for (unsigned int j = 0; j  < Mask_size; j ++) {
    Value += Tile[threadIdx.x + j] * Global_Mask[j];
  }
  Out[index] = Value;
}

*/
//:: Invocation Function

void d_convolution1d(Mat image,unsigned char *In,unsigned char *Out,char *h_Mask,int Mask_Width,int Row,int Col,int op){
  // Variables
  int Size_of_bytes =  sizeof(unsigned char)*Row*Col*image.channels();
  int Mask_size_bytes =  sizeof(char)*9;
  unsigned char *d_In, *d_Out;
  char *d_Mask;
  float Blocksize=BLOCK_SIZE;


  // Memory Allocation in device
  hipMalloc((void**)&d_In,Size_of_bytes);
  hipMalloc((void**)&d_Out,Size_of_bytes);
  hipMalloc((void**)&d_Mask,Mask_size_bytes);
  // Memcpy Host to device
  hipMemcpy(d_In,In,Size_of_bytes,hipMemcpyHostToDevice);
  hipMemcpy(d_Out,Out,Size_of_bytes,hipMemcpyHostToDevice);
  hipMemcpy(d_Mask,h_Mask,Mask_size_bytes,hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(Global_Mask),h_Mask,Mask_size_bytes); // avoid cache coherence
  // Thead logic and Kernel call
  dim3 dimGrid(ceil(Row/Blocksize),ceil(Col/Blocksize),1);
  dim3 dimBlock(Blocksize,Blocksize,1);
  if(op==1){
    convolution2d_global_kernel<<<dimGrid,dimBlock>>>(d_In,d_Mask,d_Out,Mask_Width,Row,Col);
  }
  if(op==2){
    convolution2d_constant_kernel<<<dimGrid,dimBlock>>>(d_In,d_Mask,d_Out,Mask_Width,Row,Col);
  }
  if(op==3){

  }

  hipDeviceSynchronize();
  // save output result.
  hipMemcpy (Out,d_Out,Size_of_bytes,hipMemcpyDeviceToHost);
  // Free device memory
  hipFree(d_In);
  hipFree(d_Out);
  hipFree(d_Mask);
}



// :::::::::::::::::::::::::::::::::::Clock Function::::::::::::::::::::::::::::
double diffclock(clock_t clock1,clock_t clock2){
  double diffticks=clock2-clock1;
  double diffms=(diffticks)/(CLOCKS_PER_SEC/1); // /1000 mili
  return diffms;
}
// :::::::::::::::::::::::::::::::::::::::Main::::::::::::::::::::::::::::::::::

int main(){

  int Mask_Width =  Mask_size;
  Mat image;
  image = imread("inputs/img1.jpg",0);   // Read the file
  Size s = image.size();
  int Row = s.width;
  int Col = s.height;
  //char h_Mask[] = {-1,0,1,-2,0,2,-1,0,1};
  //char h_Mask[] = {0,-1,0,-1,5,-1,0,-1,0}; Sharpen
  //char h_Mask[] = {-1,-1,-1,-1,8,-1,-1,-1,-1}; edge detection 3
  //char h_Mask[] = {0.11,0.11,0.11,0.11,0.11,0.11,0.11,0.11,0.11};
  //char h_Mask[] = {-2,-2,0,-2,6,0,0,0,0};
  //char h_Mask[] = {1,2,1,2,4,2,1,2,1}; gaussian blur
  char h_Mask[] = {-1,-1,-1,0,0,0,1,1,1}; // A kernel for edge detection
  unsigned char *img = (unsigned char*)malloc(sizeof(unsigned char)*Row*Col*image.channels());
  unsigned char *imgOut = (unsigned char*)malloc(sizeof(unsigned char)*Row*Col*image.channels());

  img = image.data;

   //::::::::::::::::::::::::::::::::::::::::: Secuential filter ::::::::::::::::::::::::::::::::::::

  /// Generate grad_x and grad_y
  //Mat grad_x, grad_y;

  /// Gradient X
  //   ( src  , grad_x, ddepth,dx,dy,scale,delta, BORDER_DEFAULT );
  //Sobel( image, grad_x, ddepth, 1, 0, 3, scale, delta, BORDER_DEFAULT );

  /// Gradient Y
  //Sobel( image, grad_y, ddepth, 0, 1, 3, scale, delta, BORDER_DEFAULT );

  //::::::::::::::::::::::::::::::::::::::::: Parallel filter ::::::::::::::::::::::::::::::::::::

  d_convolution1d(image,img,imgOut,h_Mask,Mask_Width,Row,Col,2);
  Mat gray_image;
  gray_image.create(Row,Col,CV_8UC1);
  gray_image.data = imgOut;
  imwrite("./outputs/1088015148.png",gray_image);

  //free(img);
  //free(imgOut);

  return 0;
}
/*
1 - convolution2d tile constant
2 - convolution2d notile noconstant
3 - convolution2d constant tile simple
*/
