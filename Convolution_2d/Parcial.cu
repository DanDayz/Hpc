#include "hip/hip_runtime.h"
#include <bits/stdc++.h>
#include <hip/hip_runtime.h>
#include <highgui.h>
#include <cv.h>

#define Mask_size 3
//#define TILE_size_of_rgb  1024
#define BLOCKSIZE 32

using namespace std;
using namespace cv;

__constant__ char Global_Mask[Mask_size*Mask_size];

__device__ unsigned char clamp(int value){
    if(value < 0)
        value = 0;
    else
        if(value > 255)
            value = 255;
    return  value;
}

__global__ void sobelFilter(unsigned char *In, int Row, int Col, unsigned int Mask_Width,char *Mask,unsigned char *Out){
    unsigned int row = blockIdx.y*blockDim.y+threadIdx.y;
    unsigned int col = blockIdx.x*blockDim.x+threadIdx.x;
    int Pvalue = 0;
    int N_start_point_row = row - (Mask_Width/2);
    int N_start_point_col = col - (Mask_Width/2);

    for(int i = 0; i < Mask_Width; i++){
        for(int j = 0; j < Mask_Width; j++ ){
            if((N_start_point_col + j >=0 && N_start_point_col + j < Row)&&(N_start_point_row + i >=0 && N_start_point_row + i < Col)){
                Pvalue += In[(N_start_point_row + i)*Row+(N_start_point_col + j)] * Mask[i*Mask_Width+j];
            }
        }
    }
    Out[row*Row+col] = clamp(Pvalue);
}


__global__ void sobelFilterConstant(unsigned char *In, int Row, int Col, unsigned int Mask_Width,char *Mask,unsigned char *Out){
    unsigned int row = blockIdx.y*blockDim.y+threadIdx.y;
    unsigned int col = blockIdx.x*blockDim.x+threadIdx.x;
    int Pvalue = 0;
    int N_start_point_row = row - (Mask_Width/2);
    int N_start_point_col = col - (Mask_Width/2);

    for(int i = 0; i < Mask_Width; i++){
        for(int j = 0; j < Mask_Width; j++ ){
            if((N_start_point_col + j >=0 && N_start_point_col + j < Row)&&(N_start_point_row + i >=0 && N_start_point_row + i < Col)){
                Pvalue += In[(N_start_point_row + i)*Row+(N_start_point_col + j)] * Mask[i*Mask_Width+j];
            }
        }
    }
    Out[row*Row+col] = clamp(Pvalue);
}

__global__ void gray(unsigned char *In, unsigned char *Out,int Row, int Col){
    int row = blockIdx.y*blockDim.y+threadIdx.y;
    int col = blockIdx.x*blockDim.x+threadIdx.x;

    if((row < Col) && (col < Row)){
        Out[row*Row+col] = In[(row*Row+col)*3+2]*0.299 + In[(row*Row+col)*3+1]*0.587+ In[(row*Row+col)*3+0]*0.114;
    }
}


void d_convolution2d(Mat image,unsigned char *In,unsigned char *h_Out,char *h_Mask,int Mask_Width,int Row,int Col,int op){
  // Variables
  int size_of_rgb = sizeof(unsigned char)*Row*Col*image.channels();
  int size_of_Gray = sizeof(unsigned char)*Row*Col; // sin canales alternativos
  int Mask_size_of_bytes =  sizeof(char)*(Mask_size*Mask_size);
  unsigned char *d_In,*d_Out,*d_sobelOut;
  char *d_Mask;
  float Blocksize=BLOCKSIZE;

  // Memory Allocation in device
  hipMalloc((void**)&d_In,size_of_rgb);
  hipMalloc((void**)&d_Out,size_of_Gray);
  hipMalloc((void**)&d_Mask,Mask_size_of_bytes);
  hipMalloc((void**)&d_sobelOut,size_of_Gray);

  // Memcpy Host to device
  hipMemcpy(d_In,In,size_of_rgb, hipMemcpyHostToDevice);
  hipMemcpy(d_Mask,h_Mask,Mask_size_of_bytes,hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(Global_Mask),h_Mask,Mask_size_of_bytes); // avoid cache coherence
  // Thread logic and Kernel call
  dim3 dimGrid(ceil(Row/Blocksize),ceil(Col/Blocksize),1);
  dim3 dimBlock(Blocksize,Blocksize,1);
  gray<<<dimGrid,dimBlock>>>(d_In,d_Out,Row,Col); // pasando a escala de grices.
  hipDeviceSynchronize();
  if(op==1){
    sobelFilter<<<dimGrid,dimBlock>>>(d_Out,Row,Col,Mask_size,d_Mask,d_sobelOut);
  }
  if(op==2){
    sobelFilterConstant<<<dimGrid,dimBlock>>>(d_Out,Row,Col,Mask_size,d_Mask,d_sobelOut);
  }
  if(op==3){

  }
  // save output result.
  hipMemcpy (h_Out,d_sobelOut,size_of_Gray,hipMemcpyDeviceToHost);
  // Free device memory
  hipFree(d_In);
  hipFree(d_Out);
  hipFree(d_Mask);
  hipFree(d_sobelOut);
}


int main(){

    int Mask_Width = Mask_size;
    char h_Mask[] = {-1,0,1,-2,0,2,-1,0,1};
    Mat image,result_image;
    image = imread("inputs/img2.jpg",1);
    Size s = image.size();
    int Row = s.width;
    int Col = s.height;
    unsigned char * In = (unsigned char*)malloc( sizeof(unsigned char)*Row*Col*image.channels());
    unsigned char * h_Out = (unsigned char *)malloc( sizeof(unsigned char)*Row*Col);

    In = image.data;
    d_convolution2d(image,In,h_Out,h_Mask,Mask_Width,Row,Col,1);

    result_image.create(Col,Row,CV_8UC1);
    result_image.data = h_Out;
    imwrite("./outputs/1088015148.png",result_image);


    return 0;
}
