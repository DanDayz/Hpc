#include "hip/hip_runtime.h"
#include <bits/stdc++.h>
#include <hip/hip_runtime.h>
#include <highgui.h>
#include <cv.h>

#define Mask_size 3
//#define TILE_size_of_rgb  1024
#define BLOCKSIZE 32

using namespace std;
using namespace cv;

__constant__ char Global_Mask[Mask_size*Mask_size];

__device__ unsigned char clamp(int value){
    if(value < 0)
        value = 0;
    else
        if(value > 255)
            value = 255;
    return  value;
}

__global__ void sobelFilter(unsigned char *In, int Row, int Col, unsigned int Mask_Width,char *Mask,unsigned char *Out){
    unsigned int row = blockIdx.y*blockDim.y+threadIdx.y;
    unsigned int col = blockIdx.x*blockDim.x+threadIdx.x;
    int Pvalue = 0;
    int N_start_point_row = row - (Mask_Width/2);
    int N_start_point_col = col - (Mask_Width/2);

    for(int i = 0; i < Mask_Width; i++){
        for(int j = 0; j < Mask_Width; j++ ){
            if((N_start_point_col + j >=0 && N_start_point_col + j < Row)&&(N_start_point_row + i >=0 && N_start_point_row + i < Col)){
                Pvalue += In[(N_start_point_row + i)*Row+(N_start_point_col + j)] * Mask[i*Mask_Width+j];
            }
        }
    }
    Out[row*Row+col] = clamp(Pvalue);
}


__global__ void sobelFilterConstant(unsigned char *In, int Row, int Col, unsigned int Mask_Width,char *Mask,unsigned char *Out){
    unsigned int row = blockIdx.y*blockDim.y+threadIdx.y;
    unsigned int col = blockIdx.x*blockDim.x+threadIdx.x;
    int Pvalue = 0;
    int N_start_point_row = row - (Mask_Width/2);
    int N_start_point_col = col - (Mask_Width/2);

    for(int i = 0; i < Mask_Width; i++){
        for(int j = 0; j < Mask_Width; j++ ){
            if((N_start_point_col + j >=0 && N_start_point_col + j < Row)&&(N_start_point_row + i >=0 && N_start_point_row + i < Col)){
                Pvalue += In[(N_start_point_row + i)*Row+(N_start_point_col + j)] * Mask[i*Mask_Width+j];
            }
        }
    }
    Out[row*Row+col] = clamp(Pvalue);
}


__global__ void sobelFilterShared(unsigned char *data, unsigned char *result, int width, int height){
  // Data cache: threadIdx.x , threadIdx.y
  const int n = (Mask_size*Mask_size) / 2;
  __shared__ int s_data[BLOCKSIZE + Mask_size  ][BLOCKSIZE + Mask_size ];

  // global mem address of the current thread in the whole grid
  const int pos = threadIdx.x + blockIdx.x * blockDim.x + threadIdx.y * width + blockIdx.y * blockDim.y * width;

  // load cache (32x32 shared memory, 16x16 threads blocks)
  // each threads loads four values from global memory into shared mem
  // if in image area, get value in global mem, else 0
  int x, y; // image based coordinate

  // original image based coordinate
  const int x0 = threadIdx.x + blockIdx.x * blockDim.x;
  const int y0 = threadIdx.y + blockIdx.y * blockDim.y;

  // case1: upper left
  x = x0 - n;
  y = y0 - n;
  if ( x < 0 || y < 0 )
    s_data[threadIdx.y][threadIdx.x] = 0;
  else
    s_data[threadIdx.y][threadIdx.x] = *(data + pos - n - (width * n));

  // case2: upper right
  x = x0 + n;
  y = y0 - n;
  if ( x > (width - 1) || y < 0 )
    s_data[threadIdx.y][threadIdx.x + blockDim.x] = 0;
  else
    s_data[threadIdx.y][threadIdx.x + blockDim.x] = *(data + pos + n - (width * n));

  // case3: lower left
  x = x0 - n;
  y = y0 + n;
  if (x < 0 || y > (height - 1))
    s_data[threadIdx.y + blockDim.y][threadIdx.x] = 0;
  else
    s_data[threadIdx.y + blockDim.y][threadIdx.x] = *(data + pos - n + (width * n));

  // case4: lower right
  x = x0 + n;
  y = y0 + n;
  if ( x > (width - 1) || y > (height - 1))
    s_data[threadIdx.y + blockDim.y][threadIdx.x + blockDim.x] = 0;
  else
    s_data[threadIdx.y + blockDim.y][threadIdx.x + blockDim.x] = *(data + pos + n + (width * n));

  __syncthreads();

  // convolution
  int sum = 0;
  x = n + threadIdx.x;
  y = n + threadIdx.y;
  for (int i = - n; i <= n; i++)
    for (int j = - n; j <= n; j++)
      sum += s_data[y + i][x + j] * Global_Mask[n + i] * Global_Mask[n + j];

  result[pos] = sum;
}


__global__ void gray(unsigned char *In, unsigned char *Out,int Row, int Col){
    int row = blockIdx.y*blockDim.y+threadIdx.y;
    int col = blockIdx.x*blockDim.x+threadIdx.x;

    if((row < Col) && (col < Row)){
        Out[row*Row+col] = In[(row*Row+col)*3+2]*0.299 + In[(row*Row+col)*3+1]*0.587+ In[(row*Row+col)*3]*0.114;
    }
}

// :::::::::::::::::::::::::::::::::::Clock Function::::::::::::::::::::::::::::
double diffclock(clock_t clock1,clock_t clock2){
  double diffticks=clock2-clock1;
  double diffms=(diffticks)/(CLOCKS_PER_SEC/1); // /1000 mili
  return diffms;
}

void d_convolution2d(Mat image,unsigned char *In,unsigned char *h_Out,char *h_Mask,int Mask_Width,int Row,int Col,int op){
  // Variables
  int size_of_rgb = sizeof(unsigned char)*Row*Col*image.channels();
  int size_of_Gray = sizeof(unsigned char)*Row*Col; // sin canales alternativos
  int Mask_size_of_bytes =  sizeof(char)*(Mask_size*Mask_size);
  unsigned char *d_In,*d_Out,*d_sobelOut;
  char *d_Mask;
  float Blocksize=BLOCKSIZE;

  // Memory Allocation in device
  hipMalloc((void**)&d_In,size_of_rgb);
  hipMalloc((void**)&d_Out,size_of_Gray);
  hipMalloc((void**)&d_Mask,Mask_size_of_bytes);
  hipMalloc((void**)&d_sobelOut,size_of_Gray);

  // Memcpy Host to device
  hipMemcpy(d_In,In,size_of_rgb, hipMemcpyHostToDevice);
  hipMemcpy(d_Mask,h_Mask,Mask_size_of_bytes,hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(Global_Mask),h_Mask,Mask_size_of_bytes); // avoid cache coherence
  // Thread logic and Kernel call
  dim3 dimGrid(ceil(Row/Blocksize),ceil(Col/Blocksize),1);
  dim3 dimBlock(Blocksize,Blocksize,1);
  gray<<<dimGrid,dimBlock>>>(d_In,d_Out,Row,Col); // pasando a escala de grices.
  hipDeviceSynchronize();
  if(op==1){
    sobelFilter<<<dimGrid,dimBlock>>>(d_Out,Row,Col,Mask_size,d_Mask,d_sobelOut);
  }
  if(op==2){
    sobelFilterConstant<<<dimGrid,dimBlock>>>(d_Out,Row,Col,Mask_size,d_Mask,d_sobelOut);
  }
  if(op==3){
    sobelFilterShared<<<dimGrid,dimBlock>>>(d_Out,d_sobelOut,Row,Col);
  }
  // save output result.
  hipMemcpy (h_Out,d_sobelOut,size_of_Gray,hipMemcpyDeviceToHost);
  // Free device memory
  hipFree(d_In);
  hipFree(d_Out);
  hipFree(d_Mask);
  hipFree(d_sobelOut);
}


int main(){

    double T1,T2; // Time flags
    clock_t start,end;// Time flags

    int Mask_Width = Mask_size;
    char h_Mask[] = {-1,0,1,-2,0,2,-1,0,1};
    Mat image,result_image;
    image = imread("inputs/img1.jpg",1);
    Size s = image.size();
    int Row = s.width;
    int Col = s.height;
    unsigned char * In = (unsigned char*)malloc( sizeof(unsigned char)*Row*Col*image.channels());
    unsigned char * h_Out = (unsigned char *)malloc( sizeof(unsigned char)*Row*Col);

    In = image.data;
    start = clock();
    d_convolution2d(image,In,h_Out,h_Mask,Mask_Width,Row,Col,1);
    end = clock();
    T1=diffclock(start,end);
    cout<<" Result Parallel"<<" At "<<T1<<",Seconds"<<endl;

    Mat gray_image_opencv, grad_x, abs_grad_x;
    start = clock();
    cvtColor(image, gray_image_opencv, CV_BGR2GRAY);
    Sobel(gray_image_opencv,grad_x,CV_8UC1,1,0,3,1,0,BORDER_DEFAULT);
    convertScaleAbs(grad_x, abs_grad_x);
    end = clock();
    T2=diffclock(start,end);
    cout<<" Result secuential"<<" At "<<T2<<",Seconds"<<endl;
    cout<<"Total acceleration "<<T2/T1<<"X"<<endl;

    result_image.create(Col,Row,CV_8UC1);
    result_image.data = h_Out;
    imwrite("./outputs/1088015148.png",grad_x);

    return 0;
}
