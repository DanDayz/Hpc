#include "hip/hip_runtime.h"

/* Daniel Diaz Giraldo

Restrictions
Mask = 5, Only works whit odd numbers and Mask size <= N _elements;
N_elements = defined by architecture from machine; (Femin-Maxwell....) in this case
i'm use a Kepler Arch; (the number of blocks that can support is around 2^31)

*/

#include <bits/stdc++.h>
#include <hip/hip_runtime.h>
#include <highgui.h>
#include <cv.h>

#define N_elements 32
#define Mask_size  5
#define TILE_SIZE  1024
#define BLOCK_SIZE 1024

using namespace std;
using namespace cv;


__device__ unsigned char clamp(int value){
    if(value < 0)
        value = 0;
    else
        if(value > 255)
            value = 255;
    return (unsigned char)value;
}



__global__ void convolution2d_global_kernel(unsigned char *In,unsigned char *M, unsigned char *Out,int Mask_Width,int Rowimg,int Colimg){

  
}

//:: Invocation Function

void d_convolution1d(Mat image,unsigned char *In,unsigned char *Out,char *h_Mask,unsigned int Mask_Width,unsigned int Row,unsigned int Col,int op){
  // Variables
  int Size_of_bytes =  sizeof(unsigned char)*Row*Col*image.channels();
  int Mask_size_bytes =  sizeof(unsigned char)*9;
  unsigned char *d_In, *d_Out, *d_Mask;
  float Blocksize=BLOCK_SIZE;
  
  d_In = (unsigned char*)malloc(Size_of_bytes);
  d_Out = (unsigned char*)malloc(Size_of_bytes);
  d_Mask = (unsigned char*)malloc(Mask_size_bytes);
  
  // Memory Allocation in device
  hipMalloc((void**)&d_In,Size_of_bytes);
  hipMalloc((void**)&d_Out,Size_of_bytes);
  hipMalloc((void**)&d_Mask,Mask_size_bytes);
  // Memcpy Host to device
  hipMemcpy(d_In,In,Size_of_bytes,hipMemcpyHostToDevice);
  hipMemcpy(d_Out,Out,Size_of_bytes,hipMemcpyHostToDevice);
  hipMemcpy(d_Mask,h_Mask,Mask_size_bytes,hipMemcpyHostToDevice);
  //hipMemcpyToSymbol(HIP_SYMBOL(Global_Mask),h_Mask,Mask_size*sizeof(int)); // avoid cache coherence
  // Thead logic and Kernel call
  dim3 dimGrid(ceil(Row/Blocksize),ceil(Col/Blocksize),1);
  dim3 dimBlock(ceil(Row/Blocksize),ceil(Row/Blocksize),1);
  convolution2d_global_kernel<<<dimGrid,dimBlock>>>(d_In,d_Mask,d_Out,Mask_Width,Row,Col);
  
  hipDeviceSynchronize();
  // save output result.
  hipMemcpy (Out,d_Out,Size_of_bytes,hipMemcpyDeviceToHost);
  // Free device memory
  hipFree(d_In);
  hipFree(d_Out);
  hipFree(d_Mask);
}



// :::::::::::::::::::::::::::::::::::Clock Function::::::::::::::::::::::::::::
double diffclock(clock_t clock1,clock_t clock2){
  double diffticks=clock2-clock1;
  double diffms=(diffticks)/(CLOCKS_PER_SEC/1); // /1000 mili
  return diffms;
}
// :::::::::::::::::::::::::::::::::::::::Main::::::::::::::::::::::::::::::::::

int main(){
  
  int Mask_Width =  Mask_size;
  int scale = 1;
  int delta = 0;
  int ddepth = CV_8UC1;
  Mat image;
  image = imread("inputs/img1.jpg",0);   // Read the file
  
  Size s = image.size();
  
  int Row = s.width;
  int Col = s.height;
  
  char h_Mask[] = {-1,0,1,-2,0,2,-1,0,1};
  unsigned char *img = (unsigned char*)malloc(sizeof(unsigned char)*Row*Col*image.channels());
  unsigned char *imgOut = (unsigned char*)malloc(sizeof(unsigned char)*Row*Col*image.channels());
  
  img = image.data;        
  
  
  
    
  //::::::::::::::::::::::::::::::::::::::::: Secuential filter ::::::::::::::::::::::::::::::::::::




  /// Generate grad_x and grad_y
  Mat grad_x, grad_y;

  /// Gradient X                  
  //   ( src  , grad_x, ddepth,dx,dy,scale,delta, BORDER_DEFAULT );
  Sobel( image, grad_x, ddepth, 1, 0, 3, scale, delta, BORDER_DEFAULT );

  /// Gradient Y
  //Sobel( image, grad_y, ddepth, 0, 1, 3, scale, delta, BORDER_DEFAULT );

  //
  
  //::::::::::::::::::::::::::::::::::::::::: Parallel filter ::::::::::::::::::::::::::::::::::::
    
  d_convolution1d(image,img,imgOut,h_Mask,Mask_Width,Row,Col,1);
	
  //imwrite("./outputs/1088015148.png",imgOut);
  
  //imwrite("./outputs/1088015148.png",grad_x);
  
  return 0;
}
/*
1 - convolution1d tile constant
2 - convolution1d notile noconstant
3 - convolution1d constant tile simple
*/
