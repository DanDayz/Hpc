# include <bits/stdc++.h>
# include <hip/hip_runtime.h>

#define SIZE 60// Global Size
#define BLOCK_SIZE 1024
using namespace std;

//::::::::::::::::::::::::::::::::::::::::::GPU::::::::::::::::::::::::::::::::

// :::: Kernel

__global__ void kernel_prefix_sum_inefficient(double *g_idata,double *g_odata,int l){ // Sequential Addressing technique

  __shared__ double sdata[BLOCK_SIZE];
  // each thread loads one element from global to shared mem

  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;

  if(i<l && tid !=0){ // bad thing -> severely punished performance.
    sdata[tid] = g_idata[i-1];
  }else{
    sdata[tid] = g_idata[0];
  }

  // do reduction in shared mem
  for(unsigned int s=1;s<=tid;s *=2){
    __syncthreads();
     sdata[tid]+=sdata[tid-s];
    }

  // write result for this block to global mem
  g_odata[i] = sdata[tid];
}

// :::: Calls
void d_VectorMult(double *Vec1,double *Total){
  double * d_Vec1;
  double * d_Total;
  double Blocksize=BLOCK_SIZE; // Block of 1Dim

  hipMalloc((void**)&d_Vec1,SIZE*sizeof(double));
  hipMalloc((void**)&d_Total,SIZE*sizeof(double));

  hipMemcpy(d_Vec1,Vec1,SIZE*sizeof(double),hipMemcpyHostToDevice);
  hipMemcpy(d_Total,Total,SIZE*sizeof(double),hipMemcpyHostToDevice);

  dim3 dimBlock(Blocksize,1,1);
  dim3 dimGrid(ceil(SIZE/Blocksize),1,1);
  kernel_prefix_sum_inefficient<<<dimGrid,dimBlock>>>(d_Vec1,d_Total,SIZE);

/*  int temp=SIZE;

    while(temp>1){
      dim3 dimBlock(Blocksize,1,1);
      int grid=ceil(temp/Blocksize);
      dim3 dimGrid(grid,1,1);

      KernelNormalVec
      cudaDeviceSynchronize();

      cudaMemcpy(d_Vec1,d_Total,SIZE*sizeof(double),cudaMemcpyDeviceToDevice);
      temp=ceil(temp/Blocksize);
    }
*/


    hipMemcpy(Total,d_Total,SIZE*sizeof(double),hipMemcpyDeviceToHost);
    hipFree(d_Vec1);
    hipFree(d_Total);
  }

  //::::::::::::::::::::::::::::::::::::::::::CPU::::::::::::::::::::::::::::::::

  void h_prefix_sum(double *Vec1, double *all){
    all[0]=Vec1[0];
    for(int i=0;i<SIZE;i++) all[i]=all[i-1]+Vec1[i];
  }

  //:::::::::::::::::::::::::::: Rutinary Functions

  void Fill_vec(double *Vec,double Value){
    for(int i =0 ; i<SIZE ; i++) Vec[i]=Value;
  }

  void Show_vec(double *Vec){
    for (int i=0;i<SIZE;i++){
      if(i%10==0 && i!=0){
        cout<<endl;
      }
      cout<<"["<<Vec[i]<<"] ";
    }
    cout<<endl;
  }

  void Checksum(double *Answer1 , double  *Answer2){
    if(fabs(Answer1[0]-Answer2[0]) < 0.1) cout<<"Nice Work Guy"<<endl;
    else  cout<<"BAD Work Guy"<<endl;
  }


  // :::::::::::::::::::::::::::::::::::Clock Function::::::::::::::::::::::::::::
  double diffclock(clock_t clock1,clock_t clock2){
    double diffticks=clock2-clock1;
    double diffms=(diffticks)/(CLOCKS_PER_SEC/1); // /1000 mili
    return diffms;
  }

  // :::::::::::::::::::::::::::::::::::::::Main::::::::::::::::::::::::::::::::.

int main(){

    double T1,T2; // Time flags
    double *Vec1 = (double*)malloc((SIZE)*sizeof(double)); // Elements to compute. CPU way
    double *Total2 = (double*)malloc((SIZE)*sizeof(double)); // GPU
    double *Total1 = (double*)malloc(sizeof(double)*(SIZE)); // Total Variables.

    // Fill the containers vectors of data
    Fill_vec(Vec1,1.0);
    Fill_vec(Total2,0.0);

    // Register time to finish the algorithm
    // Secuential
    clock_t start = clock();
    h_prefix_sum(Vec1,Total1);
    clock_t end = clock();
    T1=diffclock(start,end);
    Show_vec(Total1);
    //cout<<"Serial Result: "<<*Total1<<" At "<<T1<<",Seconds"<<endl;
    // Parallel

    // releasing Memory

    free(Vec1);
    free(Total1);
    free(Total2);

    return 0;
}
