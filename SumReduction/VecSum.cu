# include <bits/stdc++.h>
# include <hip/hip_runtime.h>

#define SIZE 500000// Global Size
#define TILE_WIDTH 1024
using namespace std;

// ::::::::::::::::::::::::::::::::::::::::::GPU::::::::::::::::::::::::::::::::

// :::: Kernel

__global__ void KernelNormalVec(float *g_idata,float *g_odata,int l){

         __shared__ float sdata[TILE_WIDTH];
    // each thread loads one element from global to shared mem
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
  	if(i<l){
    	sdata[tid] = g_idata[i];
    }else{
    	sdata[tid] = 0.0;
    }

    __syncthreads();
    // do reduction in shared mem
    for(unsigned int s=blockDim.x/2; s > 0; s >>= 1) {
        if(tid < s){
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }
    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

// :::: Calls
void d_VectorMult(float *Vec1,float *Total){
  float * d_Vec1;
  float * d_Total;
  float Blocksize=TILE_WIDTH; // Block of 1Dim

  hipMalloc((void**)&d_Vec1,SIZE*sizeof(float));
  hipMalloc((void**)&d_Total,SIZE*sizeof(float));

  hipMemcpy(d_Vec1, Vec1,SIZE*sizeof(float),hipMemcpyHostToDevice);
  hipMemcpy(d_Total,Total,SIZE*sizeof(float),hipMemcpyHostToDevice);

  int temp=SIZE;

  while(temp>1){
     dim3 dimBlock(Blocksize,1,1);
     int grid=ceil(temp/Blocksize);
	 	 dim3 dimGrid(grid,1,1);
     KernelNormalVec<<<dimGrid,dimBlock>>>(d_Vec1,d_Total,temp);
     hipDeviceSynchronize();
     hipMemcpy(d_Vec1,d_Total,SIZE*sizeof(float),hipMemcpyDeviceToDevice);
     temp=ceil(temp/Blocksize);
  }

  hipMemcpy(Total,d_Total,SIZE*sizeof(float),hipMemcpyDeviceToHost);

  hipFree(d_Vec1);
  hipFree(d_Total);

}

//::::::::::::::::::::::::::::::::::::::::::CPU::::::::::::::::::::::::::::::::

float h_Mul_Mat(float *Vec1){
  float all=0;
  for(int i=0;i<SIZE;i++){all+=Vec1[i];}
  return all;
}

//:::::::::::::::::::::::::::: Rutinary Functions

void Fill_vec(float *Vec,float Value){
  for(int i =0 ; i<SIZE ; i++) Vec[i]=Value;
}

void Show_vec(float *Vec){
  for (int i=0;i<SIZE;i++){
    if(i%10==0 && i!=0){
      cout<<endl;
    }
    cout<<"["<<Vec[i]<<"] ";
  }
  cout<<endl;
}

void Checksum(float Answer1 , float  *Answer2){
  if(fabs(Answer1-Answer2[0]) < 0.1) cout<<"Nice Work Guy"<<endl;
  else  cout<<"BAD Work Guy"<<endl;
}


// :::::::::::::::::::::::::::::::::::Clock Function::::::::::::::::::::::::::::
double diffclock(clock_t clock1,clock_t clock2){
  double diffticks=clock2-clock1;
  double diffms=(diffticks)/(CLOCKS_PER_SEC/1); // /1000 mili
  return diffms;
}

// :::::::::::::::::::::::::::::::::::::::Main::::::::::::::::::::::::::::::::.

int main(){

  double T1,T2; // Time flags
  float *Vec1 = (float*)malloc((SIZE)*sizeof(float)); // Elements to compute. CPU way
  // GPU "Normal" way
  float *Total2 = (float*)malloc((SIZE)*sizeof(float));
  float Total1; // Total Variables.


  // Fill the containers vectors of data
  Fill_vec(Vec1,1.0);
  Fill_vec(Total2,0.0);


  // Register time to finish the algorithm
  // Secuential
  clock_t start = clock();
  Total1=h_Mul_Mat(Vec1);
  clock_t end = clock();
  T1=diffclock(start,end);
  cout<<"Serial Result: "<<Total1<<" At "<<T1<<",Seconds"<<endl;
  // Parallel
  start = clock();
  d_VectorMult(Vec1,Total2);
  end = clock();
  T2=diffclock(start,end);
  cout<<"Parallel Result: "<<Total2[0]<<" At "<<T2<<",Seconds"<<endl;
  cout<<"Total Acceleration: "<<T1/T2<<",X"<<endl;
	Checksum(Total1,Total2);
  // releasing Memory

  free(Vec1);
  free(Total2);

  return 0;
}
