
/* Daniel Diaz Giraldo

Restrictions
Mask = 5, Only works whit odd numbers and Mask size <= N _elements;
N_elements = defined by architecture from machine; (Femin-Maxwell....) in this case
i'm use a Kepler Arch; (the number of blocks that can support is around 2^31)

*/

#include <bits/stdc++.h>
#include <hip/hip_runtime.h>

#define N_elements 7
#define Mask_size  5
#define TILE_SIZE  32

using namespace std;

__constant__ int Global_Mask[Mask_size];

//:::::::::::::::::::::::::::: Device Kernel Function ::::::::::::::::::::::::::::::

__global__ void convolution1d_tiles_constant_kernel(int *In, int *Out){
  unsigned int index = blockIdx.x * blockDim.x + threadIdx.x; // Index 1d iterator.
  __shared__ int Tile[TILE_SIZE + Mask_size - 1];
  int n = Mask_size/2;
  int halo_left_index  = (blockIdx.x - 1 )*blockDim.x + threadIdx.x;
  if (threadIdx.x  >= blockDim.x - n ){
     Tile[threadIdx.x - (blockDim.x - n )] = (halo_left_index < 0) ? 0 : In[halo_left_index];
  }
  Tile[n+threadIdx.x] = In[blockIdx.x * blockDim.x + threadIdx.x  ];

  int halo_right_index = (blockIdx.x + 1 ) * blockDim.x + threadIdx.x;
  if (threadIdx.x < n) {
    Tile[n + blockDim.x + threadIdx.x]=  (halo_left_index >= N_elements) ? 0 : In[halo_right_index];
  }
__syncthreads();
  int Value = 0;
  for (unsigned int j = 0; j  < Mask_size; j ++) {
    Value += Tile[threadIdx.x+j] * Global_Mask[j];
  }
  Out[index] = Value;
}

//:: Invocation Function

void d_convolution1d(int *In,int *Out,int *h_Mask){
  // Var
  int Size_of_bytes = N_elements * sizeof(int);
  int *d_In, *d_Out; // *d_Mask;
  float Blocksize=TILE_SIZE;
  d_In = (int*)malloc(Size_of_bytes);
  d_Out = (int*)malloc(Size_of_bytes);
  //d_Mask = (int*)malloc(Size_of_bytes);
  // Memory Allocation in device
  hipMalloc((void**)&d_In,Size_of_bytes);
  hipMalloc((void**)&d_Out,Size_of_bytes);
  //cudaMalloc((void**)&d_Mask,SIZE*sizeof(int));
  // Memcpy Host - To - device
  hipMemcpy(d_In,In,Size_of_bytes,hipMemcpyHostToDevice);
  hipMemcpy(d_Out,Out,Size_of_bytes,hipMemcpyHostToDevice);
  //cudaMemcpy(d_Mask,Mask,SIZE*sizeof(int),cudaMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(Global_Mask),h_Mask,Mask_size*sizeof(int)); // avoid cache coherence
  // Thead logic and Kernel call
  dim3 dimGrid(ceil(N_elements/Blocksize),1,1);
  dim3 dimBlock(Blocksize,1,1);
  convolution1d_tiles_constant_kernel<<<dimGrid,dimBlock>>>(d_In,d_Out);
  hipDeviceSynchronize();
  // save output result.
  hipMemcpy (Out,d_Out,Size_of_bytes,hipMemcpyDeviceToHost);
  // Free device memory
  hipFree(d_In);
  hipFree(d_Out);
  //cudaFree(d_Mask);
}

//:::::::::::::::::::::::::::: Host Function ::::::::::::::::::::::::::::::

void h_Convolution_1d(int *In,int *Out, int *Mask){
  for(int i=0;i<N_elements;i++){
    int Gap=i-(Mask_size)/2; // asymmetric Gap (Left Right)
    int Value=0;
    for(int j=0;j<Mask_size;j++){
      if(Gap+j >= 0 && j+Gap<N_elements){
        Value+=In[Gap+j]*Mask[j];
      }// end if
    }// end for j
    Out[i]=Value;
  }// end for i
}
//:::::::::::::::::::::::::::: Rutinary Functions ::::::::::::::::::::::::::::::

void Fill_elements(int * VecIn1,int Value, int n){
    for (int i = 0; i < n; i++) {
          VecIn1[i]=Value;
    }
}

void Show_vec(int *Vec,int Elements,char * Msg ){
  cout<<Msg<<endl;
  for (int i=0;i<Elements;i++){
    if(i%10==0 && i!=0){
      cout<<endl;
    }
    cout<<"["<<Vec[i]<<"] ";
  }
  cout<<endl;
}

// :::::::::::::::::::::::::::::::::::Clock Function::::::::::::::::::::::::::::
double diffclock(clock_t clock1,clock_t clock2){
  double diffticks=clock2-clock1;
  double diffms=(diffticks)/(CLOCKS_PER_SEC/1); // /1000 mili
  return diffms;
}
// :::::::::::::::::::::::::::::::::::::::Main::::::::::::::::::::::::::::::::::

int main(){

  double T1,T2; // Time flags
  clock_t start,end;// Time flags

  int *VecIn1=(int*)malloc(N_elements*sizeof(int)); // Sequential and Parallel Vector Input
  int *VecOut1=(int*)malloc(N_elements*sizeof(int)); // Sequential Vector Output
  int *VecOut2=(int*)malloc(N_elements*sizeof(int)); // Parallel Vector Output
  int *Mask=(int*)malloc(Mask_size*sizeof(int)); // Mask Vector;

  Fill_elements(VecIn1,1,N_elements);
  Fill_elements(Mask,1,Mask_size);

  Show_vec(VecIn1,N_elements,(char *)"Vector In");
  Show_vec(Mask,Mask_size,(char *)"Mask");
  start = clock();
	h_Convolution_1d(VecIn1,VecOut1,Mask);
  end = clock();
  T1=diffclock(start,end);
  cout<<"Serial Result"<<" At "<<T1<<",Seconds"<<endl;
  Show_vec(VecOut1,N_elements,(char *)"Vector Out");

  start = clock();
  d_convolution1d(VecIn1,VecOut2,Mask);
  end = clock();
  T2=diffclock(start,end);
  cout<<"Parallel Result"<<" At "<<T2<<",Seconds"<<endl;
  Show_vec(VecOut2,N_elements,(char *)"Vector Out");
  return 0;
}

/*

Book Test Values  int Mask_size = 5;
#define N_elements 7
VecIn1[0]=1;
VecIn1[1]=2;
VecIn1[2]=3;
VecIn1[3]=4;
VecIn1[4]=5;
VecIn1[5]=6;
VecIn1[6]=7;


Mask[0]=3;
Mask[1]=4;
Mask[2]=5;
Mask[3]=4;
Mask[4]=3;

*/
