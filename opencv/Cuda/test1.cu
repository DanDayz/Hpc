#include "hip/hip_runtime.h"
#include <bits/stdc++.h>

#define BLOCKSIZE 32

using namespace std;


void steganography(){

}

step_one(int height, int width , Mat image , uchar *image_data , uchar *image_example_data,string mensaje,int *lc){
   // Variables
  int size_of_rgb = sizeof(unsigned char)*Row*Col*image.channels();
  int size_of_msg = sizeof(uchar *)*mensaje.lengh();
  uchar *d_image_data,*d_image_example_data,*d_msg;
  float Blocksize=BLOCKSIZE;

  // Memory Allocation in device
  hipMalloc((void**)&d_image_data,size_of_rgb);
  hipMalloc((void**)&d_image_example_data,size_of_rgb);
  hipMalloc((void**)&d_msg,Size_of_msg);


  // Memcpy Host to device
  hipMemcpy(d_image_data,image_data,size_of_rgb, hipMemcpyHostToDevice);
  hipMemcpy(d_msg,mensaje,size_of_msg,hipMemcpyHostToDevice);
  // Thread logic and Kernel call
  dim3 dimGrid(ceil(height/Blocksize),ceil(width/Blocksize),1);
  dim3 dimBlock(Blocksize,Blocksize,1);
  gray<<<dimGrid,dimBlock>>>(d_In,d_Out,Row,Col); // pasando a escala de grices.
  hipDeviceSynchronize();
   // save output result.
  hipMemcpy (h_Out,d_sobelOut,size_of_Gray,hipMemcpyDeviceToHost);
  // Free device memory
  hipFree(d_In);
  hipFree(d_Out);
  hipFree(d_Mask);
  hipFree(d_sobelOut);

}


int main(){


  string foto="inputs/img3.jpg";
  string mensaje="hola a todos";

  cout<<"Imagen a codificar: "<<foto<<endl;

  Mat image= imread(foto,CV_LOAD_IMAGE_COLOR);
  uchar *image_data;
  Size s = image.size();
  int *lc = (unsigned int *)malloc(sizeof(unsigned int)*1);
  int width = s.width;
  int height= s.height;
  uchar *image_example_data = (unsigned char *)malloc(sizeof(unsigned char)*(width*height)*3); // 3 channels
  image_data = image.data;

  cout<<"tamaño de la imágen : "<<width <<" x "<<height<<endl;

  step_one(height,width,lc);
  //int nc=step_two(lc);
  //step_three(height,width,nc);
  //step_three(height,width,file,step_two(step_one(height,width)));
  string a="nano "+file;
  system(a.c_str());



  return 0;
}
